#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>
#include <fstream>

#include <SFML/Graphics.hpp>

#include "imgui.h"
#include <imgui-SFML.h>

//#define DOUBLE_PRECISION
#ifdef DOUBLE_PRECISION
typedef double data;
#else
typedef float data;
#endif

// Complex struct
struct Complex {
	data x, y;
	__device__ Complex() {x = 0.; y = 0.;}
	__device__ Complex(data _x, data _y) { x = _x; y = _y; }
	__device__ Complex operator+(const Complex& other) { return Complex(x + other.x, y + other.y); }
	__device__ Complex operator*(const Complex& other) { return Complex(x * other.x - y * other.y, x * other.y + y * other.x); }
};

// Typedefs and constants
typedef unsigned int uint;
typedef uint8_t u8;

const int WIDTH = 1920;
const int HEIGHT = 1024;

data cx = -0.251645; data cy = -0.768400;
data zoom = 1.0;
data expfactor = 0.0;
data max_iterations = 300.0;

// Function declarations
hipError_t populate_buffer(data* params, uint params_size, data max_iterations, uint* color_buffer);
__device__ uint encode_rgb(u8 r, u8 g, u8 b); 

// OG mandelbrot with linear grayscale coloring
__global__ void mandelbrot(data* params, data max_iterations, uint* color_buffer) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	data min_x = params[0];
	data max_x = params[1];
	data min_y = params[2];
	data max_y = params[3];

	data cx = min_x + x * (max_x - min_x) / (WIDTH - 1);
	data cy = min_y + y * (max_y - min_y) / (HEIGHT - 1);

	Complex c(cx, cy);
	Complex z = c;

	data iterations = 0; 

	while (iterations < max_iterations && z.x * z.x + z.y * z.y < 4.0) {
		z = z * z + c;
		iterations += 1.0;
	}
	uint r, g, b;
	if (iterations < max_iterations) {
		data frac = 1.0 + log2((log(16.0) / log(z.x * z.x + z.y * z.y)));

		r = (0.5 * sin(frac) + 0.5) * 255;
		g = (r * frac) / 255;
		b = frac * 255;
	} else {r = g = b = 0;}
	uint color = encode_rgb(r, g, b);

	color_buffer[x + y * WIDTH] = color;
}

__device__ data map(data x) {
	return cos(x);
}

__device__ data gamma(data x, data c, data b) {
	/*data a = 4 * pow(x, 3.0);
	data b = 4 * pow(x - 1, 3.0) + 1;
	if (x < 0.5) return a;
	return b;*/
	data col = 0.5 + (x - 0.5) * c + b;
	if (col < 0.0) return 0.0;
	else if (col > 1.0) return 1.0;
	return col;
}

// Stripe average coloring function
__global__ void mandelbrot1(data* params, data max_iterations, uint* color_buffer) {
	int x = blockIdx.x * 32 + threadIdx.x;
	int y = blockIdx.y * 32 + threadIdx.y;

	data min_x = params[0];
	data max_x = params[1];
	data min_y = params[2];
	data max_y = params[3];

	data cx = min_x + x * (max_x - min_x) / (WIDTH - 1);
	data cy = min_y + y * (max_y - min_y) / (HEIGHT - 1);

	Complex c(cx, cy);
	Complex z;

	


	Complex last_z;
	data avg = 0.0;
	data last_added = 0.0;
	data stripe_density = 5.0;
	data skip = 0;
	data count = 0;
	data escape_radius = 10000.0;

	data i = 0.0;
	while (i < max_iterations) {
		
		z = z * z + c;
		if (i >= skip) {
			count += 1.0;
			last_added = 0.5 + 0.5 * sin(stripe_density * atan2(z.y, z.x));
			avg += last_added;
		}
		
		if (z.x * z.x + z.y * z.y > escape_radius * escape_radius && i > skip) break;
		last_z = z;
		i = i + 1.0;
	}
	data prev_avg = (avg - last_added) / (count - 1.0);
	avg = avg / count;
	data frac = 1.0 + log2((log(escape_radius * escape_radius) / log(z.x * z.x + z.y * z.y)));

	data mix = frac * avg + (1.0 - frac) * prev_avg;

	uint r, g, b;
	if (i < max_iterations) {
		data dr, dg, db;
		dr = (0.5 + 0.5 * map(mix * 3.14 + params[4]));
		dg = (0.5 + 0.5 * map(mix * 3.14 + params[5]));
		db = (0.5 + 0.5 * map(mix * 3.14 + params[6]));

		b = gamma(dg, params[7], params[8]) * 255;
		g = gamma(dr, params[7], params[8]) * 255;
		r = gamma(db, params[7], params[8]) * 255;
	}
	else {
		r = g = b = 0;
	}
	uint color = encode_rgb(r, g, b);

	color_buffer[x + y * WIDTH] = color;
}

/*
	Parameters:
	0	-> Minimum X
	1	-> Maximum X
	2	-> Minimum Y
	3	-> Maximum Y
	4	-> R slider for stripe coloring
	5	-> G
	6	-> B
*/

int main() {
	data* params;
	uint* color_buffer;

	color_buffer = new uint[WIDTH * HEIGHT];
	params = new data[8];


	data aspect = WIDTH * 1.0 / HEIGHT;

	params[0] = cx - zoom * 2 * aspect;
	params[1] = cx + zoom * 2 * aspect;
	params[2] = cy - zoom * 2;
	params[3] = cy + zoom * 2;
	params[4] = -3.4;
	params[5] = 7.35;
	params[6] = 2.4;
	params[7] = 1.0;

	data movement_speed = 0.1;
	data zoom_speed = 3.;
	sf::RenderWindow window(sf::VideoMode(WIDTH, HEIGHT), "Test Window");
	sf::Image image;
	sf::Texture texture;
	sf::Sprite sprite;
	sf::Event event;
	sf::Clock deltaClock;

	bool render_new = false;

	image.create(WIDTH, HEIGHT);

	ImGui::SFML::Init(window);

	
	while (window.isOpen()) {
		while (window.pollEvent(event)) {
			ImGui::SFML::ProcessEvent(window, event);
			if (event.type == sf::Event::Closed) window.close();

			if (event.type == sf::Event::KeyPressed) {
				render_new = true;
				if (event.key.code == sf::Keyboard::W) {
					cy -= movement_speed * zoom ;
				}
				if (event.key.code == sf::Keyboard::S) {
					cy += movement_speed * zoom ;
				}
				if (event.key.code == sf::Keyboard::A) {
					cx -= movement_speed * zoom;
				}
				if (event.key.code == sf::Keyboard::D) {
					cx += movement_speed * zoom;
				}
				if (event.key.code == sf::Keyboard::LShift) max_iterations += 10;
				if (event.key.code == sf::Keyboard::LControl) max_iterations -= 10;
				if (event.key.code == sf::Keyboard::P) {
					printf("X: %lf\n", cx);
					printf("Y: %lf\n", cy);
					printf("Zoom: %lf\n", 1/zoom);
					printf("Iterations: %lf\n", max_iterations);
				}
				if (event.key.code == sf::Keyboard::O) {
					image.saveToFile("IMG.bmp");
				}

				if (event.key.code == sf::Keyboard::LBracket) expfactor += 0.02 * zoom_speed;
				if (event.key.code == sf::Keyboard::RBracket) expfactor -= 0.02 * zoom_speed;

				zoom = pow(2.0, expfactor);
				params[0] = cx - zoom * 2 * aspect;
				params[1] = cx + zoom * 2 * aspect;
				params[2] = cy - zoom * 2;
				params[3] = cy + zoom * 2;
			}
		}
		
		if (render_new) {
			populate_buffer(params, 9, max_iterations, color_buffer);
		
			for (int y = 0; y < HEIGHT; y++) for (int x = 0; x < WIDTH; ++x) {
				int idx = x + y * WIDTH;
				uint color = color_buffer[idx];
				image.setPixel(x, y, sf::Color(color));
			}
		
			texture.loadFromImage(image);
			texture.setSmooth(true);
			sprite.setTexture(texture);
		}
		window.clear();
		window.draw(sprite);

		ImGui::SFML::Update(window, deltaClock.restart());

		ImGui::Begin("Hello");
		ImGui::SetWindowFontScale(1.8);

#ifndef DOUBLE_PRECISION 
		ImGui::InputFloat("R", &params[4], 0.05f);
		ImGui::InputFloat("G", &params[5], 0.05f);
		ImGui::InputFloat("B", &params[6], 0.05f);
		ImGui::InputFloat("Contrast", &params[7], 0.05f);
		ImGui::InputFloat("Brightness", &params[8], 0.05f);
#else
		ImGui::InputDouble("R", &params[4], 0.05);
		ImGui::InputDouble("G", &params[5], 0.05);
		ImGui::InputDouble("B", &params[6], 0.05);
#endif
		ImGui::End();
		
		ImGui::SFML::Render(window);
		window.display();
		//render_new = false;
	}

	ImGui::SFML::Shutdown();

	return 0;
}


hipError_t populate_buffer(data* params, uint params_size, data max_iterations, uint* color_buffer) {
	uint* dev_color_buffer;
	data* dev_params;
	hipError_t cudaStatus;

	uint size = WIDTH * HEIGHT;

	// Allocate color buffer on the device
	cudaStatus = hipMalloc((void**)&dev_color_buffer, size * sizeof(uint));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		hipFree(dev_color_buffer);
		return cudaStatus;
	}
	
	// Allocate params on device
	cudaStatus = hipMalloc((void**)&dev_params, params_size * sizeof(data));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		hipFree(dev_params);
		hipFree(dev_color_buffer);
		return cudaStatus;
	}

	// Copy params to dev_params
	cudaStatus = hipMemcpy(dev_params, params, params_size * sizeof(data), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "params copy failed!\n");
		hipFree(dev_color_buffer);
		hipFree(dev_params);
		return cudaStatus;
	}

	dim3 threads_per_block = dim3(32, 32);
	dim3 blocks_per_grid = dim3(WIDTH / 32, HEIGHT / 32);


	mandelbrot1 <<< blocks_per_grid, threads_per_block >>> (dev_params, max_iterations, dev_color_buffer);
	
	cudaStatus = hipMemcpy(color_buffer, dev_color_buffer, size * sizeof(uint), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_color_buffer);
		return cudaStatus;
	}
	hipFree(dev_color_buffer);
	hipFree(dev_params);
}

__device__ uint encode_rgb(u8 r, u8 g, u8 b) {
	uint color = 0x00;
	color += r;
	color <<= 8;
	color += g;
	color <<= 8;
	color += b;
	color <<= 8;
	color += 0xff;

	return color;
}